#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "devcomm.h"
#include "collectives.h"
#include "common.h"

__shared__ ncclShmemData ncclShmem;

#define NCCL_FUNC5(func, algo, redop, type) \
  NCCL_FUNC_NAME(func, algo, LL,     redop, type), \
  NCCL_FUNC_NAME(func, algo, LL128,  redop, type), \
  NCCL_FUNC_NAME(func, algo, SIMPLE, redop, type)

#define NCCL_FUNC4(func, redop, type) \
  NCCL_FUNC5(func, TREE,    redop, type), \
  NCCL_FUNC5(func, RING,    redop, type), \
  NCCL_FUNC5(func, COLLNET, redop, type)

#if defined(__CUDA_BF16_TYPES_EXIST__)
// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(func, redop) \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, uint8_t), \
  NCCL_FUNC4(func, redop, int32_t), \
  NCCL_FUNC4(func, redop, uint32_t), \
  NCCL_FUNC4(func, redop, int64_t), \
  NCCL_FUNC4(func, redop, uint64_t), \
  NCCL_FUNC4(func, redop, half), \
  NCCL_FUNC4(func, redop, float), \
  NCCL_FUNC4(func, redop, double), \
  NCCL_FUNC4(func, redop, __hip_bfloat16)
#define NCCL_FUNCS3B(func, redop) \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t)
#else
// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(func, redop) \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, uint8_t), \
  NCCL_FUNC4(func, redop, int32_t), \
  NCCL_FUNC4(func, redop, uint32_t), \
  NCCL_FUNC4(func, redop, int64_t), \
  NCCL_FUNC4(func, redop, uint64_t), \
  NCCL_FUNC4(func, redop, half), \
  NCCL_FUNC4(func, redop, float), \
  NCCL_FUNC4(func, redop, double)
#define NCCL_FUNCS3B(func, redop) \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t)
#endif

// Must be consistent with ncclRedOp_t
#define NCCL_FUNCS2A(func) \
  NCCL_FUNCS3A(func, Sum ), \
  NCCL_FUNCS3A(func, Prod), \
  NCCL_FUNCS3A(func, Max ), \
  NCCL_FUNCS3A(func, Min ), \
  NCCL_FUNCS3A(func, Avg)

#define NCCL_FUNCS2B(func) \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum)

// Must be consistent with ncclFunc_t
#define NCCL_FUNCS() { \
  NCCL_FUNC_NAME(SendRecv, RING, SIMPLE, Sum, int8_t),\
  NCCL_FUNCS2B(Broadcast), \
  NCCL_FUNCS2A(Reduce), \
  NCCL_FUNCS2B(AllGather), \
  NCCL_FUNCS2A(ReduceScatter), \
  NCCL_FUNCS2A(AllReduce) }

// Must be consistent with the ncclFuncSet enum
__device__ ncclKern_t ncclFuncs[1+NCCL_NUM_FUNCTIONS*ncclNumOps*ncclNumTypes*NCCL_NUM_ALGORITHMS*NCCL_NUM_PROTOCOLS] = {
// Don't try to initialize the host shadow copy of this device-side global
// variable. There is no host pointer to a device-side function, which
// confuses clang. This will be fixed in the next clang release.
#if __CUDA_ARCH__
  NCCL_FUNC_NAME(SendRecv, RING, SIMPLE, Sum, int8_t),
  NCCL_FUNCS2B(Broadcast),
  NCCL_FUNCS2A(Reduce),
  NCCL_FUNCS2B(AllGather),
  NCCL_FUNCS2A(ReduceScatter),
  NCCL_FUNCS2A(AllReduce)
#endif
};

// Workaround for https://reviews.llvm.org/D55580
__device__ void ncclWorkaroundClangD55580() {}
